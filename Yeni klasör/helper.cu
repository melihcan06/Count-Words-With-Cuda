#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>

__global__ void assingNumberKernel(int* numbers, int size, int number) {
	int idx = threadIdx.x;//threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < size) {
		numbers[idx] = number;
	}
}

__device__ bool cmpStringKernel(char* wordArray, int firstStart, int firstEnd, int secondStart, int secondEnd){
	int firstSize = firstEnd - firstStart;
	int secondSize = secondEnd - secondStart;
	
	# if __CUDA_ARCH__>=200
		//printf("%d %d\n", secondSize, firstSize);
	#endif
	
	if (firstSize != secondSize) {
		return false;
	}
	else {
		for (int i = 0; i < firstSize; i++) {
			if (wordArray[i + firstStart] != wordArray[i + secondStart]) {
				return false;
			}
		}
		return true;
	}
}

__device__ int findWordIdxKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int startIdx) {
	if (startIdx < wordArraySize) {//en sonuncu, array in bitis idx i
		int start = wordsStartPoints[startIdx];
		int end = wordsStartPoints[startIdx + 1];
		bool cmp;

		# if __CUDA_ARCH__>=200
			//printf("%d \n", end);
		#endif

		for (int i = 0; i < startIdx; i++) {
			cmp = cmpStringKernel(wordArray, start, end, wordsStartPoints[i], wordsStartPoints[i + 1]);
			if (cmp) {
				return i;
			}
		}		
	}

	return startIdx;
}

/*
wordArray: \n olmadan uc uca ekli kelimeler
wordArraySize: kac adet kelime var bir dosyada
wordsStartPoints: kelimelerin wordArray'deki baslangic konumlari
wordCounts: sonuc icin doldurulacak olan kac kelimeden kac adetin oldugunu tutan dizi
wordArraySize tane thread baslat, wordsStartPoints un uzunlugu wordArraySize+1 dir cunku sonuncusu bitis idx'i dir.
*/
__global__ void countFileWordsKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts) {
	int idx = threadIdx.x;
	if (idx < wordArraySize) {
		int newIdx = findWordIdxKernel(wordArray, wordArraySize, wordsStartPoints, idx);		
		atomicAdd((wordCounts + newIdx), 1);
	}
}

void countFileWordsHost(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts) {
	int* wordCountsC;
	hipMalloc(&wordCountsC, sizeof(int) * wordArraySize);
	hipMemcpy(wordCountsC, wordCounts, sizeof(int) * wordArraySize, hipMemcpyHostToDevice);
	//assingNumberKernel << <1, wordArraySize >> > (wordCountsC, wordArraySize, 0);

	char* wordArrayC;
	hipMalloc(&wordArrayC, sizeof(char) * wordsStartPoints[wordArraySize]);
	hipMemcpy(wordArrayC, wordArray, sizeof(char) * wordsStartPoints[wordArraySize], hipMemcpyHostToDevice);

	int* wordsStartPointsC;
	hipMalloc(&wordsStartPointsC, sizeof(int) * (wordArraySize + 1));
	hipMemcpy(wordsStartPointsC, wordsStartPoints, sizeof(int) * (wordArraySize + 1), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(wordArraySize, 1, 1);
	countFileWordsKernel << <1, threadsPerBlock >> > (wordArrayC, wordArraySize, wordsStartPointsC, wordCountsC);

	hipMemcpy(wordCounts, wordCountsC, sizeof(int) * wordArraySize, hipMemcpyDeviceToHost);

	hipFree(wordCountsC);
	hipFree(wordArrayC);
	hipFree(wordsStartPointsC);
}

__device__ int findWordIdxForFolderKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int startIdx, int fileSize, int* fileStartPoints) {
	if (startIdx < wordArraySize) {//en sonuncu, array in bitis idx i
		int start = wordsStartPoints[startIdx];
		int end = wordsStartPoints[startIdx + 1];
		bool cmp;

		int startPointForId = 0;
		for (int i = 1; i < fileSize + 1; i++) {
			if (start < fileStartPoints[i]) {
				startPointForId = i - 1;
				break;
			}

		}
	
		int wordStartPointsStart = 0, wordStartPointsEnd = 1;
		for (int j = 0; j < wordArraySize+1; j++) {
			if (fileStartPoints[startPointForId] == wordsStartPoints[j]) {
				wordStartPointsStart = j;
			}
			if (fileStartPoints[startPointForId + 1] == wordsStartPoints[j]) {
				wordStartPointsEnd = j;
			}
		}
# if __CUDA_ARCH__>=200
		//printf("%d %d %d %d\n", startIdx, wordStartPointsStart, wordStartPointsEnd, wordArraySize);
#endif
		for (int i = wordStartPointsStart; i < wordStartPointsEnd; i++) {//startIdx
			cmp = cmpStringKernel(wordArray, start, end, wordsStartPoints[i], wordsStartPoints[i + 1]);
			if (cmp) {
			# if __CUDA_ARCH__>=200
				//printf("%d %d %d %d %d %d\n", startPointForId, startIdx, start, end, wordsStartPoints[i], wordsStartPoints[i + 1]);
			#endif
				return i;
			}
		}
	}
# if __CUDA_ARCH__>=200
	//printf("%d\n", startIdx);
#endif
	return startIdx;
}

/*
wordArray: \n olmadan uc uca ekli kelimeler
wordArraySize: kac adet kelime var bir dosyada
wordsStartPoints: kelimelerin wordArray'deki baslangic konumlari
wordCounts: sonuc icin doldurulacak olan kac kelimeden kac adetin oldugunu tutan dizi
wordArraySize tane thread baslat, wordsStartPoints un uzunlugu wordArraySize+1 dir cunku sonuncusu bitis idx'i dir.
*/
__global__ void countFolderWordsKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts, int fileSize, int* fileStartPoints) {
	int idx = threadIdx.x;
	if (idx < wordArraySize) {
		int newIdx = findWordIdxForFolderKernel(wordArray, wordArraySize, wordsStartPoints, idx, fileSize, fileStartPoints);
		atomicAdd((wordCounts + newIdx), 1);
	}
}

void countFolderWordsHost(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts, int fileSize, int* fileStartPoints) {
	int* wordCountsC;
	hipMalloc(&wordCountsC, sizeof(int) * wordArraySize);
	hipMemcpy(wordCountsC, wordCounts, sizeof(int) * wordArraySize, hipMemcpyHostToDevice);	

	char* wordArrayC;
	hipMalloc(&wordArrayC, sizeof(char) * wordsStartPoints[wordArraySize]);
	hipMemcpy(wordArrayC, wordArray, sizeof(char) * wordsStartPoints[wordArraySize], hipMemcpyHostToDevice);

	int* wordsStartPointsC;
	hipMalloc(&wordsStartPointsC, sizeof(int) * (wordArraySize + 1));
	hipMemcpy(wordsStartPointsC, wordsStartPoints, sizeof(int) * (wordArraySize + 1), hipMemcpyHostToDevice);

	int* fileStartPointsC;
	hipMalloc(&fileStartPointsC, sizeof(int) * (fileSize + 1));
	hipMemcpy(fileStartPointsC, fileStartPoints, sizeof(int) * (fileSize + 1), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(wordArraySize, 1, 1);
	countFolderWordsKernel << <1, threadsPerBlock >> > (wordArrayC, wordArraySize, wordsStartPointsC, wordCountsC, fileSize, fileStartPointsC);

	hipMemcpy(wordCounts, wordCountsC, sizeof(int) * wordArraySize, hipMemcpyDeviceToHost);

	hipFree(wordCountsC);
	hipFree(wordArrayC);
	hipFree(wordsStartPointsC);
	hipFree(fileStartPointsC);
}