#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>

__global__ void assingNumberKernel(int* numbers, int size, int number) {
	int idx = threadIdx.x;//threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < size) {
		numbers[idx] = number;
	}
}

__device__ bool cmpStringKernel(char* wordArray, int firstStart, int firstEnd, int secondStart, int secondEnd){
	int firstSize = firstEnd - firstStart;
	int secondSize = secondEnd - secondStart;
	
	# if __CUDA_ARCH__>=200
		//printf("%d %d\n", secondSize, firstSize);
	#endif
	
	if (firstSize != secondSize) {
		return false;
	}
	else {
		for (int i = 0; i < firstSize; i++) {
			if (wordArray[i + firstStart] != wordArray[i + secondStart]) {
				return false;
			}
		}
		return true;
	}
}

__device__ int findWordIdxKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int startIdx) {
	if (startIdx < wordArraySize) {//en sonuncu, array in bitis idx i
		int start = wordsStartPoints[startIdx];
		int end = wordsStartPoints[startIdx + 1];
		bool cmp;

		# if __CUDA_ARCH__>=200
			//printf("%d \n", end);
		#endif

		for (int i = 0; i < startIdx; i++) {
			cmp = cmpStringKernel(wordArray, start, end, wordsStartPoints[i], wordsStartPoints[i + 1]);
			if (cmp) {
				return i;
			}
		}		
	}

	return startIdx;
}

/*
wordArray: \n olmadan uc uca ekli kelimeler
wordArraySize: kac adet kelime var bir dosyada
wordsStartPoints: kelimelerin wordArray'deki baslangic konumlari
wordCounts: sonuc icin doldurulacak olan kac kelimeden kac adetin oldugunu tutan dizi
wordArraySize tane thread baslat, wordsStartPoints un uzunlugu wordArraySize+1 dir cunku sonuncusu bitis idx'i dir.
*/
__global__ void countFileWordsKernel(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts) {
	int idx = threadIdx.x;
	if (idx < wordArraySize) {
		int newIdx = findWordIdxKernel(wordArray, wordArraySize, wordsStartPoints, idx);		
		atomicAdd((wordCounts + newIdx), 1);
	}
}

void countFileWordsHost(char* wordArray, int wordArraySize, int* wordsStartPoints, int* wordCounts) {
	int* wordCountsC;
	hipMalloc(&wordCountsC, sizeof(int) * wordArraySize);
	hipMemcpy(wordCountsC, wordCounts, sizeof(int) * wordArraySize, hipMemcpyHostToDevice);
	//assingNumberKernel << <1, wordArraySize >> > (wordCountsC, wordArraySize, 0);

	char* wordArrayC;
	hipMalloc(&wordArrayC, sizeof(char) * wordsStartPoints[wordArraySize]);
	hipMemcpy(wordArrayC, wordArray, sizeof(char) * wordsStartPoints[wordArraySize], hipMemcpyHostToDevice);

	int* wordsStartPointsC;
	hipMalloc(&wordsStartPointsC, sizeof(int) * (wordArraySize + 1));
	hipMemcpy(wordsStartPointsC, wordsStartPoints, sizeof(int) * (wordArraySize + 1), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(wordArraySize, 1, 1);
	countFileWordsKernel << <1, threadsPerBlock >> > (wordArrayC, wordArraySize, wordsStartPointsC, wordCountsC);

	hipMemcpy(wordCounts, wordCountsC, sizeof(int) * wordArraySize, hipMemcpyDeviceToHost);

	hipFree(wordCountsC);
	hipFree(wordArrayC);
	hipFree(wordsStartPointsC);
}